#include "hip/hip_runtime.h"
#include "backend/cuda/sigmoid.cuh"

#include <hip/hip_runtime.h>

#include "backend/cuda/utils.cuh"

namespace TinyTorch::Backend::CUDA {

__global__ void sigmoid_forward_kernel(float *in, float *out, int n) {
    CUDA_KERNEL_LOOP(i, n) {
        out[i] = 1.0f / (1.0f + expf(-in[i]));
    }
}

__global__ void sigmoid_backward_kernel(float *grad_out, float *in, float *grad_in, int n) {
    CUDA_KERNEL_LOOP(i, n){
        float out = 1.0f / (1.0f + expf(-in[i]));
        grad_in[i] = grad_out[i] * out * (1 - out);
    }
}

void sigmoid_forward(float* in, float* out, int n) {
    sigmoid_forward_kernel<<<CudaGetBlocks(n), kCudaThreadsNum>>>(in, out, n);
}

void sigmoid_backward(float *grad_out, float *in, float *grad_in, int n) {
    sigmoid_backward_kernel<<<CudaGetBlocks(n), kCudaThreadsNum>>>(grad_out, in, grad_in, n);
}

}