#include "basic/mem.cuh"

#include <cstddef>

#include <hip/hip_runtime.h>

#include "basic/device.h"

namespace TinyTorch {

inline hipMemcpyKind get_cudaMemcpyKind(Device dst, Device src) {
    if (dst.is_cpu() && src.is_cpu()) {
        return hipMemcpyHostToHost;
    } else if (dst.is_cpu() && src.is_cuda()) {
        return hipMemcpyDeviceToHost;
    } else if (dst.is_cuda() && src.is_cpu()) {
        return hipMemcpyHostToDevice;
    } else if (dst.is_cuda() && src.is_cuda()) {
        return hipMemcpyDeviceToDevice;
    } else {}
}

void memcpy(void *dst_ptr, Device dst, void *src_ptr, Device src, size_t length) {
    hipMemcpy(dst_ptr, src_ptr, length, get_cudaMemcpyKind(dst, src));
}

}